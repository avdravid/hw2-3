#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define NUM_THREADS 256


double binSize;
int binNum;
particle_t* d_particles = nullptr;
particle_t* d_temp_particles = nullptr;
int* d_bin_counts = nullptr;
int* d_bin_starts = nullptr;
int* d_bin_offsets = nullptr;
void* d_temp_storage = nullptr;
size_t temp_storage_bytes = 0;

#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;

    if (r2 > cutoff * cutoff || r2 == 0) return;

    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);
    double coef = (1.0 - cutoff / r) / r2 / mass;

    particle.ax += coef * dx;
    particle.ay += coef * dy;
}


__global__ void compute_bin_indices_kernel(particle_t* particles, int* counts, int n, double binSize, int binNum) {
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n; tid += blockDim.x * gridDim.x) {
        if (tid < n) {
            particle_t p = particles[tid];
            int bin_x = min(binNum - 1, max(0, int(p.x / binSize)));
            int bin_y = min(binNum - 1, max(0, int(p.y / binSize)));
            int bin_idx = bin_y * binNum + bin_x;
            atomicAdd(&counts[bin_idx], 1);
        }
    }
}

__global__ void sort_particles_kernel(particle_t* particles, particle_t* sorted, int* bin_starts, 
                                     int* bin_offsets, int n, double binSize, int binNum) {
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n; tid += blockDim.x * gridDim.x) {
        if (tid < n) {
            particle_t p = particles[tid];
            int bin_x = min(binNum - 1, max(0, int(p.x / binSize)));
            int bin_y = min(binNum - 1, max(0, int(p.y / binSize)));
            int bin_idx = bin_y * binNum + bin_x;
            int dest_idx = bin_starts[bin_idx] + atomicAdd(&bin_offsets[bin_idx], 1);
            sorted[dest_idx] = p;
        }
    }
}


__global__ void compute_forces_and_move_gpu(particle_t* binned_particles, particle_t* original_particles,
                                            int* bin_starts, int* bin_counts, int n, double binSize, 
                                            int binNum, double size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n) return;

    particle_t* p = &original_particles[tid];
    p->ax = 0.0;
    p->ay = 0.0;

    int bin_x = min(binNum - 1, max(0, int(p->x / binSize)));
    int bin_y = min(binNum - 1, max(0, int(p->y / binSize)));


    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            int nx = bin_x + dx;
            int ny = bin_y + dy;
            if (nx < 0 || nx >= binNum || ny < 0 || ny >= binNum) continue;
            int bin_idx = ny * binNum + nx;
            int start = bin_starts[bin_idx];
            int count = bin_counts[bin_idx];
            for (int j = 0; j < count; j++) {
                particle_t neighbor = binned_particles[start + j];
                apply_force_gpu(*p, neighbor);
            }
        }
    }


    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;


    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}


void init_simulation(particle_t* parts_gpu, int num_parts, double size) {
    binSize = cutoff;
    binNum = max(1, int(size / binSize) + 1);

    d_particles = parts_gpu;
    CUDA_CHECK(hipMalloc(&d_temp_particles, num_parts * sizeof(particle_t)));
    CUDA_CHECK(hipMalloc(&d_bin_counts, binNum * binNum * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_bin_starts, binNum * binNum * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_bin_offsets, binNum * binNum * sizeof(int)));

    CUDA_CHECK(hipMemset(d_bin_counts, 0, binNum * binNum * sizeof(int)));
    CUDA_CHECK(hipMemset(d_bin_starts, 0, binNum * binNum * sizeof(int)));
    CUDA_CHECK(hipMemset(d_bin_offsets, 0, binNum * binNum * sizeof(int)));

    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_bin_counts, d_bin_starts, binNum * binNum);
    CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
}

void simulate_one_step(particle_t* parts_gpu, int num_parts, double size) {
    int blocks = min(108 * 4, (num_parts + NUM_THREADS - 1) / NUM_THREADS); 

    CUDA_CHECK(hipMemset(d_bin_counts, 0, binNum * binNum * sizeof(int)));
    CUDA_CHECK(hipMemset(d_bin_offsets, 0, binNum * binNum * sizeof(int)));

    compute_bin_indices_kernel<<<blocks, NUM_THREADS>>>(d_particles, d_bin_counts, num_parts, binSize, binNum);
    CUDA_CHECK(hipGetLastError());

    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_bin_counts, d_bin_starts, binNum * binNum);
    CUDA_CHECK(hipGetLastError());

    sort_particles_kernel<<<blocks, NUM_THREADS>>>(d_particles, d_temp_particles, d_bin_starts, d_bin_offsets, num_parts, binSize, binNum);
    CUDA_CHECK(hipGetLastError());

    compute_forces_and_move_gpu<<<blocks, NUM_THREADS>>>(d_temp_particles, d_particles, d_bin_starts, d_bin_counts, num_parts, binSize, binNum, size);
    CUDA_CHECK(hipGetLastError());
}

void free_simulation_resources() {
    if (d_temp_particles) hipFree(d_temp_particles);
    if (d_bin_counts) hipFree(d_bin_counts);
    if (d_bin_starts) hipFree(d_bin_starts);
    if (d_bin_offsets) hipFree(d_bin_offsets);
    if (d_temp_storage) hipFree(d_temp_storage);

    d_temp_particles = nullptr;
    d_bin_counts = nullptr;
    d_bin_starts = nullptr;
    d_bin_offsets = nullptr;
    d_temp_storage = nullptr;
}
